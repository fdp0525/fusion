#include "hip/hip_runtime.h"
#include "gpu_sdf.h"

#include <hip/hip_runtime.h>

__global__ void 
data_energy_kernel(float * x, int w, int h, int d){
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= w * h * d){
        return;
    }
}

__global__ void 
killing_energy_kernel(float * x, int w, int h, int d){
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= w * h * d){
        return;
    }
}

__global__ void 
level_set_energy_kernel(float * x, int w, int h, int d){
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= w * h * d){
        return;
    }
}

gpu_sdf_t::gpu_sdf_t(depth_map_t depths, point_t size, float l) : sdf_t(depths, size, l, true){

}
